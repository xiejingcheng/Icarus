#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include "cudarray/common.hpp"
#include "cudarray/array_ops.hpp"


const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;

namespace cudarray {


template<typename T, unsigned int axis>
__global__ void kernel_concatenate(
    const T *a, const T *b, unsigned int d0, unsigned int d1, unsigned int d2,
    unsigned int da, unsigned int db, T *c
) {
  CUDA_GRID_STRIDE_LOOP(c_idx, d0*d1*d2) {
    unsigned int c2_idx = c_idx % d2;
    unsigned int c1_idx = (c_idx / d2) % d1;
    unsigned int c0_idx = c_idx / d2 / d1;
    unsigned int a_idx;
    unsigned int b_idx;
    bool from_a;
    if (axis == 0) {
      a_idx = (c0_idx*d1 + c1_idx)*d2 + c2_idx;
      b_idx = ((c0_idx-da)*d1 + c1_idx)*d2 + c2_idx;
      from_a = c0_idx < da;
    }
    if (axis == 1) {
      a_idx = (c0_idx*da + c1_idx)*d2 + c2_idx;
      b_idx = (c0_idx*db + (c1_idx-da))*d2 + c2_idx;
      from_a = c1_idx < da;
    }
    if (axis == 2) {
      a_idx = (c0_idx*d1 + c1_idx)*da + c2_idx;
      b_idx = (c0_idx*d1 + c1_idx)*db + (c2_idx-da);
      from_a = c2_idx < da;
    }
    c[c_idx] = from_a ? a[a_idx] : b[b_idx];
  }
}

template<typename T>
void concatenate(
    const T *a, const T *b, unsigned int axis, unsigned int d0,
    unsigned int d1, unsigned int d2, unsigned int da, unsigned int db, T *c
) {
  unsigned int n = d0*d1*d2;
  if (axis == 0) {
    kernel_concatenate<T, 0><<<cuda_blocks(n), kNumBlockThreads>>>(
      a, b, d0, d1, d2, da, db, c
    );
  } else if (axis == 1) {
    kernel_concatenate<T, 1><<<cuda_blocks(n), kNumBlockThreads>>>(
      a, b, d0, d1, d2, da, db, c
    );
  } else if (axis == 2) {
    kernel_concatenate<T, 2><<<cuda_blocks(n), kNumBlockThreads>>>(
      a, b, d0, d1, d2, da, db, c
    );
  } else {
    throw std::runtime_error("invalid axis");
  }
  CUDA_KERNEL_CHECK;
}

template void concatenate<float>(
    const float *a, const float *b, unsigned int axis, unsigned int d0,
    unsigned int d1, unsigned int d2, unsigned int da, unsigned int db,
    float *c
);
template void concatenate<int>(
    const int *a, const int *b, unsigned int axis, unsigned int d0,
    unsigned int d1, unsigned int d2, unsigned int da, unsigned int db,
    int *c
);


template<typename T, unsigned int axis>
__global__ void kernel_split(
    const T *c, unsigned int d0, unsigned int d1, unsigned int d2,
    unsigned int da, unsigned int db, T *a, T *b
) {
  CUDA_GRID_STRIDE_LOOP(c_idx, d0*d1*d2) {
    unsigned int c2_idx = c_idx % d2;
    unsigned int c1_idx = (c_idx / d2) % d1;
    unsigned int c0_idx = c_idx / d2 / d1;
    unsigned int a_idx;
    unsigned int b_idx;
    bool from_a;
    if (axis == 0) {
      a_idx = (c0_idx*d1 + c1_idx)*d2 + c2_idx;
      b_idx = ((c0_idx-da)*d1 + c1_idx)*d2 + c2_idx;
      from_a = c0_idx < da;
    }
    if (axis == 1) {
      a_idx = (c0_idx*da + c1_idx)*d2 + c2_idx;
      b_idx = (c0_idx*db + (c1_idx-da))*d2 + c2_idx;
      from_a = c1_idx < da;
    }
    if (axis == 2) {
      a_idx = (c0_idx*d1 + c1_idx)*da + c2_idx;
      b_idx = (c0_idx*d1 + c1_idx)*db + (c2_idx-da);
      from_a = c2_idx < da;
    }
    T val = c[c_idx];
    if (from_a) {
        a[a_idx] = val;
    } else {
        b[b_idx] = val;
    }
  }
}


template<typename T>
void split(
    const T *c, unsigned int axis, unsigned int d0, unsigned int d1,
    unsigned int d2, unsigned int da, unsigned int db, T *a, T *b
) {
  unsigned int n = d0*d1*d2;
  if (axis == 0) {
    kernel_split<T, 0><<<cuda_blocks(n), kNumBlockThreads>>>(
      c, d0, d1, d2, da, db, a, b
    );
  } else if (axis == 1) {
    kernel_split<T, 1><<<cuda_blocks(n), kNumBlockThreads>>>(
      c, d0, d1, d2, da, db, a, b
    );
  } else if (axis == 2) {
    kernel_split<T, 2><<<cuda_blocks(n), kNumBlockThreads>>>(
      c, d0, d1, d2, da, db, a, b
    );
  } else {
    throw std::runtime_error("invalid axis");
  }
  CUDA_KERNEL_CHECK;
}

template void split<float>(
    const float *c, unsigned int axis, unsigned int d0, unsigned int d1,
    unsigned int d2, unsigned int da, unsigned int db, float *a, float *b
);
template void split<int>(
    const int *c, unsigned int axis, unsigned int d0, unsigned int d1,
    unsigned int d2, unsigned int da, unsigned int db, int *a, int *b
);


// Adapted from
// http://devblogs.nvidia.com/parallelforall/efficient-matrix-transpose-cuda-cc/
template<typename T, bool mTileMultiple, bool nTileMultiple>
__global__ void kernel_transpose(const T *a, unsigned int m, unsigned int n,
                                 T *b) {
  __shared__ T tile[TILE_DIM][TILE_DIM+1];

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  for (int i = 0; i < TILE_DIM; i += blockDim.y) {
    int y_ = y + i;
    if (mTileMultiple || y_ < m) {
      if (nTileMultiple || x < n) {
        tile[threadIdx.y + i][threadIdx.x] = a[y_*n + x];
      }
    }
  }
  __syncthreads();

  x = blockIdx.y * blockDim.x + threadIdx.x;
  y = blockIdx.x * TILE_DIM + threadIdx.y;
  for (int i = 0; i < TILE_DIM; i += blockDim.y) {
    int y_ = y + i;
    if (nTileMultiple || y_ < n) {
      if (mTileMultiple || x < m) {
        b[y_*m + x] = tile[threadIdx.x][threadIdx.y + i];
      }
    }
  }
}


#define ceildiv(a, b) (((a)+(b)-1)/(b))

template<typename T>
void transpose(const T *a, unsigned int m, unsigned int n, T *b) {
  dim3 blocks(ceildiv(n,TILE_DIM), ceildiv(m,TILE_DIM), 1);
  dim3 threads(TILE_DIM, BLOCK_ROWS, 1);
  if (m % TILE_DIM) {
    if (n % TILE_DIM) {
      kernel_transpose<T, false, false><<<blocks, threads>>>(a, m, n, b);
    } else {
      kernel_transpose<T, false, true><<<blocks, threads>>>(a, m, n, b);
    }
  } else {
    if (n % TILE_DIM) {
      kernel_transpose<T, true, false><<<blocks, threads>>>(a, m, n, b);
    } else {
      kernel_transpose<T, true, true><<<blocks, threads>>>(a, m, n, b);
    }
  }
  CUDA_KERNEL_CHECK;
}

template void transpose<int>(const int *a, unsigned int m, unsigned int n,
                             int *b);
template void transpose<float>(const float *a, unsigned int m, unsigned int n,
                               float *b);


template<typename Ta, typename Tb>
__global__ void kernel_as(const Ta *a, unsigned int n, Tb *b) {
  CUDA_GRID_STRIDE_LOOP(idx, n) {
    b[idx] = (Tb) a[idx];
  }
}

template<typename Ta, typename Tb>
void as(const Ta *a, unsigned int n, Tb *b) {
  kernel_as<Ta, Tb><<<cuda_blocks(n), kNumBlockThreads>>>(a, n, b);
  CUDA_KERNEL_CHECK;
}

template void as<int, float>(const int *a, unsigned int n, float *b);
template void as<float, int>(const float *a, unsigned int n, int *b);


template<typename T>
__global__ void kernel_fill(T *a, unsigned int n, T alpha) {
  CUDA_GRID_STRIDE_LOOP(idx, n) {
    a[idx] = alpha;
  }
}

template<typename T>
void fill(T *a, unsigned int n, T alpha) {
  kernel_fill<T><<<cuda_blocks(n), kNumBlockThreads>>>(a, n, alpha);
  CUDA_KERNEL_CHECK;
}

template void fill<int>(int *a, unsigned int n, int alpha);
template void fill<float>(float *a, unsigned int n, float alpha);


template<typename T>
void copy(const T *a, unsigned int n, T *b) {
  CUDA_CHECK(hipMemcpy(b, a, n*sizeof(T), hipMemcpyDeviceToDevice));
}

template void copy<int>(const int *a, unsigned int n, int *b);
template void copy<float>(const float *a, unsigned int n, float *b);


template<typename T>
void to_device(const T *a, unsigned int n, T *b) {
  CUDA_CHECK(hipMemcpy(b, a, n*sizeof(T), hipMemcpyHostToDevice));
}

template void to_device<int>(const int *a, unsigned int n, int *b);
template void to_device<float>(const float *a, unsigned int n, float *b);


template<typename T>
void to_host(const T *a, unsigned int n, T *b) {
  CUDA_CHECK(hipMemcpy(b, a, n*sizeof(T), hipMemcpyDeviceToHost));
}

template void to_host<int>(const int *a, unsigned int n, int *b);
template void to_host<float>(const float *a, unsigned int n, float *b);

}
