//https://blog.csdn.net/tiger1334/article/details/44703317

#include "hip/hip_runtime.h"

#include "hip/device_functions.h"
#include <stdio.h>
#include <stdlib.h>
#include<assert.h>
#define SHARED_SIZE_LIMIT 1024U
#define    UMUL(a, b) __umul24((a), (b))
#define UMAD(a, b, c) ( UMUL((a), (b)) + (c) )
typedef unsigned int uint;
 
__device__ inline void Comparator(
    uint &keyA,
    uint &valA,
    uint &keyB,
    uint &valB,
    uint dir
)
{
    uint t;
 
    if ((keyA > keyB) == dir)
    {
        t = keyA;
        keyA = keyB;
        keyB = t;
        t = valA;
        valA = valB;
        valB = t;
    }
}
__global__ void bitonicSortShared(
    uint *d_DstKey,
    uint *d_DstVal,
    uint *d_SrcKey,
    uint *d_SrcVal,
    uint arrayLength,
    uint dir
)
{
    //Shared memory storage for one or more short vectors
    __shared__ uint s_key[SHARED_SIZE_LIMIT];
    __shared__ uint s_val[SHARED_SIZE_LIMIT];
 
    //Offset to the beginning of subbatch and load data
    d_SrcKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_SrcVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    s_key[threadIdx.x +                       0] = d_SrcKey[                      0];
    s_val[threadIdx.x +                       0] = d_SrcVal[                      0];
    s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcKey[(SHARED_SIZE_LIMIT / 2)];
    s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcVal[(SHARED_SIZE_LIMIT / 2)];
 
    for (uint size = 2; size < arrayLength; size <<= 1)
    {
        //Bitonic merge
        uint ddd = dir ^ ((threadIdx.x & (size / 2)) != 0);
 
        for (uint stride = size / 2; stride > 0; stride >>= 1)
        {
            __syncthreads();
            uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            Comparator(
                s_key[pos +      0], s_val[pos +      0],
                s_key[pos + stride], s_val[pos + stride],
                ddd
            );
        }
    }
 
    //ddd == dir for the last bitonic merge step
    {
        for (uint stride = arrayLength / 2; stride > 0; stride >>= 1)
        {
            __syncthreads();
            uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            Comparator(
                s_key[pos +      0], s_val[pos +      0],
                s_key[pos + stride], s_val[pos + stride],
                dir
            );
        }
    }
 
    __syncthreads();
    d_DstKey[                      0] = s_key[threadIdx.x +                       0];
    d_DstVal[                      0] = s_val[threadIdx.x +                       0];
    d_DstKey[(SHARED_SIZE_LIMIT / 2)] = s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
    d_DstVal[(SHARED_SIZE_LIMIT / 2)] = s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}
 
 
 
 
// Bitonic sort kernel for large arrays (not fitting into shared memory)
 
//Bottom-level bitonic sort
//Almost the same as bitonicSortShared with the exception of
//even / odd subarrays being sorted in opposite directions
//Bitonic merge accepts both
//Ascending | descending or descending | ascending sorted pairs
__global__ void bitonicSortShared1(
    uint *d_DstKey,
    uint *d_DstVal,
    uint *d_SrcKey,
    uint *d_SrcVal
)
{
    //Shared memory storage for current subarray
    __shared__ uint s_key[SHARED_SIZE_LIMIT];
    __shared__ uint s_val[SHARED_SIZE_LIMIT];
 
    //Offset to the beginning of subarray and load data
    d_SrcKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_SrcVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    s_key[threadIdx.x +                       0] = d_SrcKey[                      0];
    s_val[threadIdx.x +                       0] = d_SrcVal[                      0];
    s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcKey[(SHARED_SIZE_LIMIT / 2)];
    s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcVal[(SHARED_SIZE_LIMIT / 2)];
 
    for (uint size = 2; size < SHARED_SIZE_LIMIT; size <<= 1)
    {
        //Bitonic merge
        uint ddd = (threadIdx.x & (size / 2)) != 0;
 
        for (uint stride = size / 2; stride > 0; stride >>= 1)
        {
            __syncthreads();
            uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            Comparator(
                s_key[pos +      0], s_val[pos +      0],
                s_key[pos + stride], s_val[pos + stride],
                ddd
            );
        }
    }
 
    //Odd / even arrays of SHARED_SIZE_LIMIT elements
    //sorted in opposite directions
    uint ddd = blockIdx.x & 1;
    {
        for (uint stride = SHARED_SIZE_LIMIT / 2; stride > 0; stride >>= 1)
        {
            __syncthreads();
            uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            Comparator(
                s_key[pos +      0], s_val[pos +      0],
                s_key[pos + stride], s_val[pos + stride],
                ddd
            );
        }
    }
 
 
    __syncthreads();
    d_DstKey[                      0] = s_key[threadIdx.x +                       0];
    d_DstVal[                      0] = s_val[threadIdx.x +                       0];
    d_DstKey[(SHARED_SIZE_LIMIT / 2)] = s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
    d_DstVal[(SHARED_SIZE_LIMIT / 2)] = s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}
 
//Bitonic merge iteration for stride >= SHARED_SIZE_LIMIT
__global__ void bitonicMergeGlobal(
    uint *d_DstKey,
    uint *d_DstVal,
    uint *d_SrcKey,
    uint *d_SrcVal,
    uint arrayLength,
    uint size,
    uint stride,
    uint dir
)
{
    uint global_comparatorI = blockIdx.x * blockDim.x + threadIdx.x;
    uint        comparatorI = global_comparatorI & (arrayLength / 2 - 1);
 
    //Bitonic merge
    uint ddd = dir ^ ((comparatorI & (size / 2)) != 0);
    uint pos = 2 * global_comparatorI - (global_comparatorI & (stride - 1));
 
    uint keyA = d_SrcKey[pos +      0];
    uint valA = d_SrcVal[pos +      0];
    uint keyB = d_SrcKey[pos + stride];
    uint valB = d_SrcVal[pos + stride];
 
    Comparator(
        keyA, valA,
        keyB, valB,
        ddd
    );
 
    d_DstKey[pos +      0] = keyA;
    d_DstVal[pos +      0] = valA;
    d_DstKey[pos + stride] = keyB;
    d_DstVal[pos + stride] = valB;
}
 
//Combined bitonic merge steps for
//size > SHARED_SIZE_LIMIT and stride = [1 .. SHARED_SIZE_LIMIT / 2]
__global__ void bitonicMergeShared(
    uint *d_DstKey,
    uint *d_DstVal,
    uint *d_SrcKey,
    uint *d_SrcVal,
    uint arrayLength,
    uint size,
    uint dir
)
{
    //Shared memory storage for current subarray
    __shared__ uint s_key[SHARED_SIZE_LIMIT];
    __shared__ uint s_val[SHARED_SIZE_LIMIT];
 
    d_SrcKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_SrcVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    s_key[threadIdx.x +                       0] = d_SrcKey[                      0];
    s_val[threadIdx.x +                       0] = d_SrcVal[                      0];
    s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcKey[(SHARED_SIZE_LIMIT / 2)];
    s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcVal[(SHARED_SIZE_LIMIT / 2)];
 
    //Bitonic merge
    uint comparatorI = UMAD(blockIdx.x, blockDim.x, threadIdx.x) & ((arrayLength / 2) - 1);
    uint ddd = dir ^ ((comparatorI & (size / 2)) != 0);
 
    for (uint stride = SHARED_SIZE_LIMIT / 2; stride > 0; stride >>= 1)
    {
        __syncthreads();
        uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
        Comparator(
            s_key[pos +      0], s_val[pos +      0],
            s_key[pos + stride], s_val[pos + stride],
            ddd
        );
    }
 
    __syncthreads();
    d_DstKey[                      0] = s_key[threadIdx.x +                       0];
    d_DstVal[                      0] = s_val[threadIdx.x +                       0];
    d_DstKey[(SHARED_SIZE_LIMIT / 2)] = s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
    d_DstVal[(SHARED_SIZE_LIMIT / 2)] = s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}
 
 
 
 
// Interface function
 
//Helper function (also used by odd-even merge sort)
extern "C" uint factorRadix2(uint *log2L, uint L)
{
    if (!L)
    {
        *log2L = 0;
        return 0;
    }
    else
    {
        for (*log2L = 0; (L & 1) == 0; L >>= 1, *log2L++);
 
        return L;
    }
}
 
extern "C" uint bitonicSort(
    uint *d_DstKey,
    uint *d_DstVal,
    uint *d_SrcKey,
    uint *d_SrcVal,
    uint batchSize,
    uint arrayLength,
    uint dir
)
{
    //Nothing to sort
    if (arrayLength < 2)
        return 0;
 
    //Only power-of-two array lengths are supported by this implementation
    uint log2L;
    uint factorizationRemainder = factorRadix2(&log2L, arrayLength);
    assert(factorizationRemainder == 1);
 
    dir = (dir != 0);
 
    uint  blockCount = batchSize * arrayLength / SHARED_SIZE_LIMIT;
    uint threadCount = SHARED_SIZE_LIMIT / 2;
 
    if (arrayLength <= SHARED_SIZE_LIMIT)
    {
        assert((batchSize * arrayLength) % SHARED_SIZE_LIMIT == 0);
        bitonicSortShared<<<blockCount, threadCount>>>(d_DstKey, d_DstVal, d_SrcKey, d_SrcVal, arrayLength, dir);
    }
    else
    {
        bitonicSortShared1<<<blockCount, threadCount>>>(d_DstKey, d_DstVal, d_SrcKey, d_SrcVal);
 
        for (uint size = 2 * SHARED_SIZE_LIMIT; size <= arrayLength; size <<= 1)
            for (unsigned stride = size / 2; stride > 0; stride >>= 1)
                if (stride >= SHARED_SIZE_LIMIT)
                {
                    bitonicMergeGlobal<<<(batchSize * arrayLength) / 512, 256>>>(d_DstKey, d_DstVal, d_DstKey, d_DstVal, arrayLength, size, stride, dir);
                }
                else
                {
                    bitonicMergeShared<<<blockCount, threadCount>>>(d_DstKey, d_DstVal, d_DstKey, d_DstVal, arrayLength, size, dir);
                    break;
                }
    }
 
    return threadCount;
}
int main(){
const uint             N = 1048576;
    const uint           DIR = 0;
    const uint     numValues = 65536;
 
uint *h_InputKey, *h_InputVal, *h_OutputKeyGPU, *h_OutputValGPU;
    uint *d_InputKey, *d_InputVal,    *d_OutputKey,    *d_OutputVal;
h_InputKey     = (uint *)malloc(N * sizeof(uint));
    h_InputVal     = (uint *)malloc(N * sizeof(uint));
    h_OutputKeyGPU = (uint *)malloc(N * sizeof(uint));
    h_OutputValGPU = (uint *)malloc(N * sizeof(uint));
 
	   for (uint i = 0; i < N; i++)
    {
        h_InputKey[i] = rand() % numValues;
        h_InputVal[i] = i;
    }
	 hipMalloc((void **)&d_InputKey,  N * sizeof(uint));
 hipMalloc((void **)&d_InputVal,  N * sizeof(uint));
 hipMalloc((void **)&d_OutputKey, N * sizeof(uint));
 hipMalloc((void **)&d_OutputVal, N * sizeof(uint));
 hipMemcpy(d_InputKey, h_InputKey, N * sizeof(uint), hipMemcpyHostToDevice);
 hipMemcpy(d_InputVal, h_InputVal, N * sizeof(uint), hipMemcpyHostToDevice);
	   
	   int flag = 1;
	   for(uint arrayLength = 64;arrayLength<=N;arrayLength*=2){
		   hipDeviceSynchronize();
		   bitonicSort(d_OutputKey,d_OutputVal,d_InputKey,d_InputVal,
			   N/arrayLength,arrayLength,DIR);
		   hipDeviceSynchronize();
	   }
	    hipMemcpy(h_OutputKeyGPU, d_OutputKey, N * sizeof(uint), hipMemcpyDeviceToHost);
		hipMemcpy(h_OutputValGPU, d_OutputVal, N * sizeof(uint), hipMemcpyDeviceToHost);
		for(int i=0;i<N;i++){
		printf("%d %d\n",h_OutputKeyGPU[i],h_OutputValGPU[i]);
		}
		 hipFree(d_OutputVal);
         hipFree(d_OutputKey);
         hipFree(d_InputVal);
         hipFree(d_InputKey);
		 free(h_OutputValGPU);
		 free(h_OutputKeyGPU);
		 free(h_InputVal);
		 free(h_InputKey);
		 system("pause");
		 return 0;
}