#include "random.cuh"
#include <hiprand.h>


EXPORT int random_normal(matrix *a, float mu, float sigma, unsigned int n) {
  hiprandGenerator_t generator;
  hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT); 
  int len = a->size[0] * a->size[1];
  hiprandGenerateUniform(generator, a->data_device, len);
  return 0;
}