#include "random.cuh"
#include <hiprand.h>


EXPORT int random_normal(tensor *a, float mu, float sigma, unsigned int n) {
  hiprandGenerator_t generator;
  hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT); 
  int len = 1;
  for(int dim = 0; dim < a->dims; dim++){
    len *= a->size[dim];
  }
  hiprandGenerateUniform(generator, a->data_device, len);
  return 0;
}