#include <stdio.h>
#include <errno.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils/device.cuh"
#include "operator/gemm.cuh"
#include "data_error.cuh"
#include "oi/print.cuh"

extern "C" {

    //检查最后一个错误，并且打印错误信息
    inline bool checkCUDAError() {
        hipError_t err = hipGetLastError();
    
        if (hipSuccess != err)
            printf("%s\n", hipGetErrorString( err));
        return hipSuccess != err;
    }

    EXPORT const char* get_last_cuda_error() {
        hipError_t err = hipGetLastError();
    
        return hipGetErrorString( err);
    }

    EXPORT const char* get_last_clib_error() {
        return strerror(errno);
    }

    EXPORT int cuda_set_device(int deviceId) {
        hipSetDevice(deviceId);
        
        if (checkCUDAError())
            return CUDA_ERROR;
        else
            return 0;
    }

    EXPORT void set_transpose(tensor* data, int is_trans) {
        data->is_trans = is_trans;
    }
    
    inline char get_transpose_char(tensor* data) {
        return data->is_trans ? 't' : 'n';
    }
    
    EXPORT void cuda_sync_threads() {
        hipDeviceSynchronize();
    }

    EXPORT int allocate_device_memory(tensor* data) {
        int len = 1;
    
        for(int dim = 0; dim < data->dims; dim++){
            len *= data->size[dim];
        }
    
        hipError_t cuda_status = hipMalloc((void**)&data->data_device, len * sizeof(data->data_device[0]));
    
        if (cuda_status != hipSuccess || checkCUDAError()) {
            checkCUDAError();
            return CUDA_ERROR;
        }
    
        data->on_device = 1;
        return 0;
    }
    
    EXPORT int copy_to_host(tensor* data) {
        int len = 1;
    
        for(int dim = 0; dim < data->dims; dim++){
            len *= data->size[dim];
        }
    
        if (data->on_device) {
            hipError_t cuda_status = hipMemcpy(data->data_host, data->data_device, len * sizeof(data->data_host[0]), hipMemcpyDeviceToHost);
    
            if (cuda_status != hipSuccess || checkCUDAError()) {
                checkCUDAError();
                return CUDA_ERROR;
            }
        } else {
            return ERROR_NOT_ON_DEVICE;
        }
    
     
        return 0;
    }
    
    EXPORT int copy_to_device(tensor* data) {
        int len = 1;
    
        for(int dim = 0; dim < data->dims; dim++){
            len *= data->size[dim];
        }
    
        int err_code = 0;
    

        if (!data->on_device) {
            err_code = allocate_device_memory(data);
            if (err_code)
                return err_code;
        }
    
        hipError_t cuda_status = hipMemcpy(data->data_device, data->data_host, len * sizeof(data->data_host[0]), hipMemcpyHostToDevice);
    
        if (cuda_status != hipSuccess || checkCUDAError()) {
            checkCUDAError();
            return CUDA_ERROR;
        }

    
        return 0;
    }
    
    EXPORT int copy_on_device(tensor* data1, tensor* data2) {
        int len = 1;
    
        for(int dim = 0; dim < data1->dims; dim++){
            len *= data1->size[dim];
        }
    
        for(int dim = 0; dim < data1->dims; dim++){
            if (data1->size[dim] != data2->size[dim])
                return ERROR_INCOMPATIBLE_DIMENSIONS;
        }
        
        hipError_t cuda_status = hipMemcpy(data2->data_device, data1->data_device, len * sizeof(data1->data_device[0]), hipMemcpyDeviceToDevice);
    
        if (cuda_status != hipSuccess || checkCUDAError()) {
            checkCUDAError();
            return CUDA_ERROR;
        }
        else
            return 0;
    }
    
    EXPORT int free_device_memory(tensor* data) {
        if (data->on_device) {
            hipError_t cuda_status = hipFree(data->data_device);
            data->on_device = 0;
    
            if (cuda_status != hipSuccess || checkCUDAError()) {
                checkCUDAError();
                return CUDA_ERROR;
            }
        }
    
        return 0;
    }

    EXPORT int build_matrix_empty(int m, int n, tensor* data) {
        data->dims = 2;
        data->size[0] = m;
        data->size[1] = n;
        data->data_host = (float*)malloc(m * n * sizeof(float));
        data->owns_data = 1;
        data->on_device = 0;
        data->on_host = 1;
        return 0;
    }

    EXPORT int build_tensor_empty(int dims, int* size, tensor* data) {
        data->dims = dims;
        for(int i = 0; i < dims; i++){
            data->size[i] = size[i];
        }
        int len = 1;
        for(int i = 0; i < dims; i++){
            len *= size[i];
        }
        data->data_host = (float*)malloc(len * sizeof(float));
        data->owns_data = 1;
        data->on_device = 0;
        data->on_host = 1;
        return 0;
    }


    EXPORT int gemm_on_device(tensor* data1, tensor* data2, tensor* result) {
        if (data1->dims==2 && data1->dims==2){
            gemm_on_device_2d(data1, data2, result);
            return 0;
        }
        else{
            printf("Currently only support matrix multiplication.");
            return ERROR_INCOMLETE;
        }
    }
    


}